
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <cstdint>
#include <iostream>

#define DEVICE_ORDINAL  (0)
#define THREADS_PER_BLK (256)
#define LEN             (65520 * 1024 * 2)
#define STAGES          (128)
#define REPS            (16)
#define ITER            (10)

const int DEPTH = STAGES;

#if defined(_WIN32)
#if !defined(WIN32_LEAN_AND_MEAN)
#define WIN32_LEAN_AND_MEAN
#endif
#include <windows.h>
double second (void)
{
    LARGE_INTEGER t;
    static double oofreq;
    static int checkedForHighResTimer;
    static BOOL hasHighResTimer;

    if (!checkedForHighResTimer) {
        hasHighResTimer = QueryPerformanceFrequency (&t);
        oofreq = 1.0 / (double)t.QuadPart;
        checkedForHighResTimer = 1;
    }
    if (hasHighResTimer) {
        QueryPerformanceCounter (&t);
        return (double)t.QuadPart * oofreq;
    } else {
        return (double)GetTickCount() * 1.0e-3;
    }
}
#elif defined(__linux__) || defined(__APPLE__)
#include <stddef.h>
#include <sys/time.h>
double second (void)
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double)tv.tv_sec + (double)tv.tv_usec * 1.0e-6;
}
#else
#error unsupported platform
#endif

// Macro to catch CUDA errors in CUDA runtime calls
#define CUDA_SAFE_CALL(call)                                          \
do {                                                                  \
    hipError_t err = call;                                           \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err) );       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)

// Macro to catch CUDA errors in kernel launches
#define CHECK_LAUNCH_ERROR()                                          \
do {                                                                  \
    /* Check synchronous errors, i.e. pre-launch */                   \
    hipError_t err = hipGetLastError();                             \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err) );       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
    /* Check asynchronous errors, i.e. kernel failed (ULF) */         \
    err = hipDeviceSynchronize();                                    \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString( err) );      \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)

template <typename T>
__device__ T mad_mix (T a, T b, T c)
{
    c = a * b + c;
    a = b * c + a;
    b = c * a + b;
    return b;
}

template <typename T>
__global__ void kernel (const T * __restrict__ src, 
                        T * __restrict__ dst, 
                        T a, T b, int len)
{
    int stride = gridDim.x * blockDim.x;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    T aa = __sinf(a) * a;
    T bb = __cosf(b) * b;
    T cc = __sinf(b) * b;
    T dd = __cosf(a) * a;
    for (int i = tid; i < len; i += stride) {
        T p = src[i] * aa + bb;
        T q = src[i] * bb + aa;
        T r = src[i] * cc + dd;
        T s = src[i] * dd + cc;
        for (int k = 0; k < REPS; k++) {
#pragma unroll DEPTH
            for (int j = 0; j < DEPTH; j++) {
                p = mad_mix(p, bb, aa);
                q = mad_mix(q, bb, aa);
                r = mad_mix(r, bb, aa);
                s = mad_mix(s, bb, aa);
            }
        }
        dst[i] = p * q * r * s;
    }
}    

int main (void)
{
    double start, stop, elapsed, mintime, nbr_of_mad;
    uint32_t *d_a, *d_b;
    struct hipDeviceProp_t props;

    CUDA_SAFE_CALL (hipGetDeviceProperties (&props, DEVICE_ORDINAL));
    printf ("running on device %d (%s)\n", DEVICE_ORDINAL, props.name);

    /* Allocate memory on device */
    CUDA_SAFE_CALL (hipMalloc((void**)&d_a, sizeof(d_a[0]) * LEN));
    CUDA_SAFE_CALL (hipMalloc((void**)&d_b, sizeof(d_b[0]) * LEN));
    
    /* Initialize device memory */
    CUDA_SAFE_CALL (hipMemset(d_a, 0x00, sizeof(d_a[0]) * LEN)); // zero

    /* Compute execution configuration */
    dim3 dimBlock(THREADS_PER_BLK);
    int threadBlocks = (LEN + (dimBlock.x - 1)) / dimBlock.x;

    dim3 dimGrid(threadBlocks);
    
    printf ("using %d threads per block, %d blocks, %f GB used\n", 
            dimBlock.x, dimGrid.x, 2*1e-9*LEN*sizeof(d_a[0]));

    nbr_of_mad = (DEPTH * REPS * 12.0 + 4.0 + 3.0) * LEN;
    
    printf ("testing INT32 op throughput with IMAD (one IMAD = two iops)\n");
    mintime=1e308;
    for (int k = 0; k < ITER; k++) {
        hipDeviceSynchronize();
        start = second();
        kernel<uint32_t><<<dimGrid,dimBlock>>>(d_a, d_b, 0x5da07326, 0x5102d832, LEN);
        CHECK_LAUNCH_ERROR();
        stop = second();
        elapsed= stop - start;
        if (elapsed < mintime) { mintime = elapsed; }
    }
    printf ("iops=%13.6e  elapsed=%.5f sec  throughput=%.5f Tiops (via IMAD)\n",
            nbr_of_mad * 2, mintime, nbr_of_mad * 2 * 1e-12 / mintime);

    printf ("testing FP32 op throughput with FMAD (one FMAD = two flops)\n");
    mintime=1e308;
    for (int k = 0; k < ITER; k++) {
        hipDeviceSynchronize();
        start = second();
        kernel<float><<<dimGrid,dimBlock>>>((float*)d_a, (float*)d_b, 0x5da07326, 0x5102d832, LEN);
        CHECK_LAUNCH_ERROR();
        stop = second();
        elapsed= stop - start;
        if (elapsed < mintime) { mintime = elapsed; }
    }
    printf ("flops=%13.6e  elapsed=%.5f sec  throughput=%.5f Tflops (via FMAD)\n",
            nbr_of_mad * 2, mintime, nbr_of_mad * 2 * 1e-12 / mintime);

    CUDA_SAFE_CALL (hipFree(d_a));
    CUDA_SAFE_CALL (hipFree(d_b));

    return EXIT_SUCCESS;
}